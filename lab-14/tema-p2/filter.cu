#include "hip/hip_runtime.h"

#include <cstdlib>
#include <fstream>
#include <iostream>

using namespace std;

//  b,g,r,b,g,r,b,g,r
// [0,1,2,3,4,5,6,7,8]

__global__ void applyFilterOnGPU(const int *inputPixelArray, int *outputPixelArray, int lines, int cols, int channels) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < lines * cols * channels) {
        if (id % 3 != 0) {
            // BGR representation!
            int inputBlue = inputPixelArray[id];
            int inputGreen = inputPixelArray[id + 1];
            int inputRed = inputPixelArray[id + 2];
            int blueValue = 0;
            int greenValue = 0;
            int redValue = 0;

            // should be sepia
            blueValue = (inputRed * .272) + (inputGreen *.534) + (inputBlue * .131);
            greenValue = (inputRed * .349) + (inputGreen *.686) + (inputBlue * .168);
            redValue = (inputRed * .393) + (inputGreen *.769) + (inputBlue * .189);

            if (blueValue > 255)
                blueValue = 255;
            if (greenValue > 255)
                greenValue = 255;
            if (redValue > 255)
                redValue = 255;

            outputPixelArray[id] = blueValue;
            outputPixelArray[id + 1] = greenValue;
            outputPixelArray[id + 2] = redValue;
        }
    }
}

int *flatten(int ***matrix, int lines, int cols, int channels) {
    int *flat = (int *) malloc(lines * cols * channels * sizeof(int));
    int id = 0;
    for (int i = 0; i < lines; i++) {
        for (int j = 0; j < cols; j++) {
            for (int c = 0; c < channels; c++) {
                flat[id] = matrix[i][j][c];
                id++;
            }
        }
    }
    return flat;
}

int ***unflatten(const int *array, int lines, int cols, int channels) {
    int ***img = (int ***) malloc(lines * sizeof(int **));
    int id = 0;

    for (int i = 0; i < lines; i++) {
        img[i] = (int **) malloc(cols * sizeof(int *));

        for (int j = 0; j < cols; j++) {
            img[i][j] = (int *) malloc(channels * sizeof(int));

            for (int c = 0; c < channels; c++) {
                img[i][j][c] = array[id];
                id++;
            }
        }
    }
    return img;
}

void applyFilter() {
    //read the matrix
    ifstream in("pixels.txt");
    int lines, cols, channels = 3;
    in >> lines >> cols;

    int BLOCK_SIZE = 1000;
    int blockCount = ((lines * cols * channels) / BLOCK_SIZE) + 1;

    //matrix read
    int ***matrix = (int ***) malloc(lines * sizeof(int **));
    for (int i = 0; i < lines; i++) {
        matrix[i] = (int **) malloc(cols * sizeof(int *));

        for (int j = 0; j < cols; j++) {
            int *line = (int *) malloc(channels * sizeof(int));

            in >> line[0] >> line[1] >> line[2];

            matrix[i][j] = line;
        }
    }

    int size = lines * cols * channels;
    //flatten matrix
    int *flatMatrix = flatten(matrix, lines, cols, channels);
    int *result = (int *) malloc(size * sizeof(int));

    //copy flatMatrix to the device array
    int *deviceMatrix, *deviceResult;
    hipMalloc(&deviceMatrix, size * sizeof(int));
    hipMalloc(&deviceResult, size * sizeof(int));

    hipMemcpy(deviceMatrix, flatMatrix, size * sizeof(int), hipMemcpyHostToDevice);

    //call the filter function
    applyFilterOnGPU <<< blockCount, 1000 >>>(deviceMatrix, deviceResult, lines, cols, channels);

    //copy from deviceResult in result
    hipMemcpy(result, deviceResult, size * sizeof(int), hipMemcpyDeviceToHost);

    int ***img = unflatten(result, lines, cols, channels);

    ofstream out("pixels.txt");
    out << lines << " " << cols << " " << channels << "\n";
    for (int i = 0; i < lines; i++) {
        for (int j = 0; j < cols; j++) {
            for (int k = 0; k < channels; k++) {
                out << img[i][j][k] << " ";
            }
            out << "\n";
        }
    }

    out.close();
}

int main() {
    char *inCommand = "python ../img_to_px.py ../img.jpg";
    char *outCommand = "python ../px_to_img.py ../sepia.jpg";

    cout << "converting image to pixel values matrix" << endl;
    system(inCommand); //read pixel values to pixels.txt
    cout << "applying filter" << endl;
    applyFilter();
    cout << "converting pixel values matrix to image" << endl;
    system(outCommand); //write pixel values to image

    return 0;
}
